#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>

// type alias

template <typename scalar_t, int dims>
using PackedAccessor = torch::PackedTensorAccessor32<scalar_t, dims, torch::RestrictPtrTraits>;

// forward kernel

template <typename scalar_t>
__global__ void forward_kernel(
    const PackedAccessor<scalar_t, 4> q,
    const PackedAccessor<scalar_t, 4> k,
    const PackedAccessor<scalar_t, 4> v,
          PackedAccessor<scalar_t, 4> o,
          PackedAccessor<scalar_t, 3> l,
    const float scale,
    const int q_block_size,
    const int k_block_size
) {
    const int batch_idx = blockIdx.x;
    const int head_idx = blockIdx.y;

    const int q_seq_len = q.size(-2);
    const int k_seq_len = k.size(-2);
    const int dim = q.size(-1);

    const int num_col_tiles = (k_seq_len + k_block_size - 1) / k_block_size;
    const int num_row_tiles = (q_seq_len + q_block_size - 1) / q_block_size;

    const int row_tile_idx = threadIdx.x;
    const int col_tile_idx = threadIdx.y;

    int col_tiles_offset, row_tiles_offset;

    for (int i = 0; i < num_col_tiles; i++) {
        col_tiles_offset = i * k_block_size;

        for (int j = 0; j < num_row_tiles; j++) {
            row_tiles_offset = j * q_block_size;
        }
    }
}

 // backward kernel

template <typename scalar_t>
__global__ void backward_kernel(
    const PackedAccessor<scalar_t, 4> q,
    const PackedAccessor<scalar_t, 4> k,
    const PackedAccessor<scalar_t, 4> v,
          PackedAccessor<scalar_t, 4> dq,
          PackedAccessor<scalar_t, 4> dk,
          PackedAccessor<scalar_t, 4> dv,
    const PackedAccessor<scalar_t, 4> grad_o,
    const PackedAccessor<scalar_t, 4> o,
    const PackedAccessor<scalar_t, 3> l,
    const float scale,
    const int q_block_size,
    const int k_block_size
) {
    const int batch_idx = blockIdx.x;
    const int head_idx = blockIdx.y;

    const int q_seq_len = q.size(-2);
    const int k_seq_len = k.size(-2);
    const int dim = q.size(-1);

    const int num_col_tiles = (k_seq_len + k_block_size - 1) / k_block_size;
    const int num_row_tiles = (q_seq_len + q_block_size - 1) / q_block_size;

    const int row_tile_idx = threadIdx.x;
    const int col_tile_idx = threadIdx.y;

    int col_tiles_offset, row_tiles_offset;

    for (int i = 0; i < num_col_tiles; i++) {
        col_tiles_offset = i * k_block_size;

        for (int j = 0; j < num_row_tiles; j++) {
            row_tiles_offset = j * q_block_size;
        }
    }
}

// main c++ function

std::vector<torch::Tensor> flash_cosine_sim_attention_forward(
    torch::Tensor q,
    torch::Tensor k,
    torch::Tensor v,
    float scale,
    int q_block_size,
    int k_block_size
) {
    auto o = torch::zeros_like(q);
    const at::cuda::OptionalCUDAGuard device_guard(device_of(o));

    auto l = torch::zeros_like(q).sum({-1,});

    const int batch = q.size(0);
    const int heads = q.size(1);

    const dim3 threads_per_block(q_block_size, k_block_size);
    const dim3 blocks(batch, heads);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(q.scalar_type(), "forward_cosine_sim_attention_forward", ([&] {
        forward_kernel<scalar_t><<<blocks, threads_per_block>>>(
            q.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            k.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            v.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            o.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            l.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            scale,
            q_block_size,
            k_block_size
        );
    }));

    hipDeviceSynchronize();

    // handle error

    hipError_t error = hipGetLastError();

    if(error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // output

    return {o, l};
}

std::vector<torch::Tensor> flash_cosine_sim_attention_backward(
    torch::Tensor grad_o,
    torch::Tensor o,
    torch::Tensor l,
    torch::Tensor q,
    torch::Tensor k,
    torch::Tensor v,
    float scale,
    int q_block_size,
    int k_block_size
) {
    auto dq = torch::zeros_like(q);
    auto dk = torch::zeros_like(k);
    auto dv = torch::zeros_like(v);

    const at::cuda::OptionalCUDAGuard device_guard(device_of(dq));

    const int batch = dq.size(0);
    const int heads = dq.size(1);

    const dim3 threads_per_block(q_block_size, k_block_size);
    const dim3 blocks(batch, heads);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(q.scalar_type(), "forward_cosine_sim_attention_backward", ([&] {
        backward_kernel<scalar_t><<<blocks, threads_per_block>>>(
            q.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            k.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            v.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            dq.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            dk.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            dv.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            grad_o.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            o.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
            l.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
            scale,
            q_block_size,
            k_block_size
        );
    }));

    hipDeviceSynchronize();

    // handle error

    hipError_t error = hipGetLastError();

    if(error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // output

    return {dq, dk, dv};
}

// bind

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &flash_cosine_sim_attention_forward, "Flash Cosine-Sim Attention Forward");
    m.def("backward", &flash_cosine_sim_attention_backward, "Flash Cosine-Sim Attention Backward");
}
